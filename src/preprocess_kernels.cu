#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2021 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <iostream>

#include <hip/hip_runtime_api.h>

#include "preprocess_kernels.h"

#define checkCudaErrors(status)                                   \
{                                                                 \
  if (status != 0)                                                \
  {                                                               \
    std::cout << "Cuda failure: " << hipGetErrorString(status)   \
              << " at line " << __LINE__                          \
              << " in file " << __FILE__                          \
              << " error status: " << status                      \
              << std::endl;                                       \
              abort();                                            \
    }                                                             \
}

//<<PreProcessCuda::generateVoxels
__global__ void generateVoxels_random_kernel(float *points, size_t points_size,
        float min_x_range, float max_x_range,
        float min_y_range, float max_y_range,
        float min_z_range, float max_z_range,
        float pillar_x_size, float pillar_y_size, float pillar_z_size,
        int grid_y_size, int grid_x_size,
        unsigned int *mask, float *voxels)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(point_idx >= points_size) return;


  float4 point = ((float4*)points)[point_idx];

  if(point.x<min_x_range||point.x>=max_x_range
    || point.y<min_y_range||point.y>=max_y_range
    || point.z<min_z_range||point.z>=max_z_range) return;

  int voxel_idx = floorf((point.x - min_x_range)/pillar_x_size);
  int voxel_idy = floorf((point.y - min_y_range)/pillar_y_size);
  unsigned int voxel_index = voxel_idy * grid_x_size
                            + voxel_idx;

  unsigned int point_id = atomicAdd(&(mask[voxel_index]), 1);

  if(point_id >= POINTS_PER_VOXEL) return;
  float *address = voxels + (voxel_index*POINTS_PER_VOXEL + point_id)*4;
  atomicExch(address+0, point.x);
  atomicExch(address+1, point.y);
  atomicExch(address+2, point.z);
  atomicExch(address+3, point.w);
}

hipError_t generateVoxels_random_launch(float *points, size_t points_size,
        float min_x_range, float max_x_range,
        float min_y_range, float max_y_range,
        float min_z_range, float max_z_range,
        float pillar_x_size, float pillar_y_size, float pillar_z_size,
        int grid_y_size, int grid_x_size,
        unsigned int *mask, float *voxels,
        hipStream_t stream)
{
  int threadNum = THREADS_FOR_VOXEL;
  dim3 blocks((points_size+threadNum-1)/threadNum);
  dim3 threads(threadNum);
  generateVoxels_random_kernel<<<blocks, threads, 0, stream>>>
    (points, points_size,
        min_x_range, max_x_range,
        min_y_range, max_y_range,
        min_z_range, max_z_range,
        pillar_x_size, pillar_y_size, pillar_z_size,
        grid_y_size, grid_x_size,
        mask, voxels);
  hipError_t err = hipGetLastError();
  return err;
}

//<<PreProcessCuda::generateVoxels
__global__ void generateVoxelsList_kernel(float *points, size_t points_size,
        float min_x_range, float max_x_range,
        float min_y_range, float max_y_range,
        float min_z_range, float max_z_range,
        float pillar_x_size, float pillar_y_size, float pillar_z_size,
        int grid_y_size, int grid_x_size,
        unsigned int *mask, int *voxelsList)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(point_idx >= points_size) return;

  float4 point = ((float4*)points)[point_idx];

  if(point.x<min_x_range||point.x>=max_x_range
    || point.y<min_y_range||point.y>=max_y_range
    || point.z<min_z_range||point.z>=max_z_range)
  {
    voxelsList[point_idx] = -1;
    return;
  }

  int voxel_idx = floorf((point.x - min_x_range)/pillar_x_size);
  int voxel_idy = floorf((point.y - min_y_range)/pillar_y_size);
  unsigned int voxel_index = voxel_idy * grid_x_size
                            + voxel_idx;

  atomicAdd(&(mask[voxel_index]), 1);
  voxelsList[point_idx] = voxel_index;

}

void generateVoxelsList_cpu(float *points, size_t points_size,
        unsigned int *mask, int *voxelsList)
{
  hipDeviceSynchronize();
  for(int point_idx = points_size-1; point_idx>=0; point_idx--)
  {
    //float4 point = ((float4*)points)[point_idx];
    int voxel_index = voxelsList[point_idx];
    if(voxel_index ==-1) continue;
    int count = mask[voxel_index];
    //printf("idx:%d voxel_index %d count: %d\n",point_idx, voxel_index, count);
    if(count>32)
    {
      voxelsList[point_idx] = -1;
      mask[voxel_index]--;
      continue;
    }

    //clear mask buffer
    if(count==0) continue;
    if(count<=32)
    {
      mask[voxel_index]=0;
      continue;
    }
  }
  return;
}

__global__ void generateVoxels_kernel(float *points, size_t points_size,
        int *voxelsList,
        unsigned int *mask, float *voxels)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(point_idx >= points_size) return;

  int voxel_index = voxelsList[point_idx];

  if (voxel_index == -1) return;
  int point_id = atomicAdd(&(mask[voxel_index]), 1);

  if(point_id >= POINTS_PER_VOXEL) return;
  float *address = voxels + (voxel_index*POINTS_PER_VOXEL + point_id)*4;
  float4 point = ((float4*)points)[point_idx];
  atomicExch(address+0, point.x);
  atomicExch(address+1, point.y);
  atomicExch(address+2, point.z);
  atomicExch(address+3, point.w);
}

__global__ void generateBaseFeatures_kernel(unsigned int *mask, float *voxels,
        int grid_y_size, int grid_x_size,
        unsigned int *pillar_num,
        float *voxel_features,
        float *voxel_num_points,
        float *coords)
{
  int voxel_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int voxel_idy = blockIdx.y * blockDim.y + threadIdx.y;

  if(voxel_idx >= grid_x_size ||voxel_idy >= grid_y_size) return;

  unsigned int voxel_index = voxel_idy * grid_x_size
                           + voxel_idx;
  unsigned int count = mask[voxel_index];
  if( !(count>0) ) return;
  count = count<POINTS_PER_VOXEL?count:POINTS_PER_VOXEL;

  unsigned int current_pillarId = 0;
  current_pillarId = atomicAdd(pillar_num+4, 1);

  voxel_num_points[current_pillarId] = count;

  float4 coord = {0.0, 0, (float)voxel_idy, (float)voxel_idx};
  ((float4*)coords)[current_pillarId] = coord;

  for (int i=0; i<count; i++){
    int inIndex = voxel_index*POINTS_PER_VOXEL + i;
    int outIndex = current_pillarId*POINTS_PER_VOXEL + i;
    ((float4*)voxel_features)[outIndex] = ((float4*)voxels)[inIndex];
  }

  // clear buffer for next infer
  //mask[voxel_index] = 0;
  atomicExch(mask + voxel_index, 0);
}

hipError_t generateVoxels_launch(float *points, size_t points_size,
        float min_x_range, float max_x_range,
        float min_y_range, float max_y_range,
        float min_z_range, float max_z_range,
        float pillar_x_size, float pillar_y_size, float pillar_z_size,
        int grid_y_size, int grid_x_size,
        unsigned int *mask, float *voxels, int *voxelsList,
        hipStream_t stream)
{
  int threadNum = THREADS_FOR_VOXEL;
  dim3 blocks((points_size+threadNum-1)/threadNum);
  dim3 threads(threadNum);
  generateVoxelsList_kernel<<<blocks, threads, 0, stream>>>
    (points, points_size,
        min_x_range, max_x_range,
        min_y_range, max_y_range,
        min_z_range, max_z_range,
        pillar_x_size, pillar_y_size, pillar_z_size,
        grid_y_size, grid_x_size,
        mask, voxelsList);

  generateVoxelsList_cpu(points, points_size,
        mask, voxelsList);

  generateVoxels_kernel<<<blocks, threads, 0, stream>>>
    (points, points_size,
        voxelsList,
        mask, voxels);
  hipError_t err = hipGetLastError();
  return err;
}

/* create 4 channels*/
hipError_t generateBaseFeatures_launch(unsigned int *mask, float *voxels,
        int grid_y_size, int grid_x_size,
        unsigned int *pillar_num,
        float *voxel_features,
        float *voxel_num_points,
        float *coords,
        hipStream_t stream)
{
  dim3 threads = {32,32};
  dim3 blocks = {(grid_x_size + threads.x -1)/threads.x,
                 (grid_y_size + threads.y -1)/threads.y};

  generateBaseFeatures_kernel<<<blocks, threads, 0, stream>>>
      (mask, voxels, grid_y_size, grid_x_size,
       pillar_num,
       voxel_features,
       voxel_num_points,
       coords);
  hipError_t err = hipGetLastError();
  return err;
}
//PreProcessCuda::generateVoxels>>


//<<generateFeatures 4 channels -> 10 channels
__global__ void generateFeatures_kernel(float* voxel_features,
    float* voxel_num_points, float* coords, unsigned int *params,
    float voxel_x, float voxel_y, float voxel_z,
    float range_min_x, float range_min_y, float range_min_z,
    float* features)
{
    int pillar_idx = blockIdx.x * WARPS_PER_BLOCK + threadIdx.x/WARP_SIZE;
    int point_idx = threadIdx.x % WARP_SIZE;

    int pillar_idx_inBlock = threadIdx.x/32;
    unsigned int num_pillars = params[4];

    if (pillar_idx >= num_pillars) return;

    //load src
    __shared__ float4 pillarSM[WARPS_PER_BLOCK][WARP_SIZE]; //4*32*4
    __shared__ float4 pillarSumSM[WARPS_PER_BLOCK]; //4*4
    __shared__ float4 cordsSM[WARPS_PER_BLOCK]; //4*4
    __shared__ int pointsNumSM[WARPS_PER_BLOCK]; //4
    __shared__ float pillarOutSM[WARPS_PER_BLOCK][WARP_SIZE][FEATURES_SIZE]; //4*32*10

    if (threadIdx.x < WARPS_PER_BLOCK) {
      pointsNumSM[threadIdx.x] = voxel_num_points[blockIdx.x * WARPS_PER_BLOCK + threadIdx.x];
      cordsSM[threadIdx.x] = ((float4*)coords)[blockIdx.x * WARPS_PER_BLOCK + threadIdx.x];
      pillarSumSM[threadIdx.x] = {0,0,0,0};
    }

    pillarSM[pillar_idx_inBlock][point_idx] = ((float4*)voxel_features)[pillar_idx*WARP_SIZE + point_idx];
    __syncthreads();

    //calculate sm in a pillar
    if (point_idx < pointsNumSM[pillar_idx_inBlock]) {
      atomicAdd(&(pillarSumSM[pillar_idx_inBlock].x),  pillarSM[pillar_idx_inBlock][point_idx].x);
      atomicAdd(&(pillarSumSM[pillar_idx_inBlock].y),  pillarSM[pillar_idx_inBlock][point_idx].y);
      atomicAdd(&(pillarSumSM[pillar_idx_inBlock].z),  pillarSM[pillar_idx_inBlock][point_idx].z);
    }
    __syncthreads();

    //feature-mean
    float4 mean;
    float validPoints = pointsNumSM[pillar_idx_inBlock];
    mean.x = pillarSumSM[pillar_idx_inBlock].x / validPoints;
    mean.y = pillarSumSM[pillar_idx_inBlock].y / validPoints;
    mean.z = pillarSumSM[pillar_idx_inBlock].z / validPoints;

    mean.x  = pillarSM[pillar_idx_inBlock][point_idx].x - mean.x;
    mean.y  = pillarSM[pillar_idx_inBlock][point_idx].y - mean.y;
    mean.z  = pillarSM[pillar_idx_inBlock][point_idx].z - mean.z;


    //calculate offset
    float x_offset = voxel_x / 2 + cordsSM[pillar_idx_inBlock].w * voxel_x + range_min_x;
    float y_offset = voxel_y / 2 + cordsSM[pillar_idx_inBlock].z * voxel_y + range_min_y;
    float z_offset = voxel_z / 2 + cordsSM[pillar_idx_inBlock].y * voxel_z + range_min_z;

    //feature-offset
    float4 center;
    center.x  = pillarSM[pillar_idx_inBlock][point_idx].x - x_offset;
    center.y  = pillarSM[pillar_idx_inBlock][point_idx].y - y_offset;
    center.z  = pillarSM[pillar_idx_inBlock][point_idx].z - z_offset;

    //store output
    if (point_idx < pointsNumSM[pillar_idx_inBlock]) {
      pillarOutSM[pillar_idx_inBlock][point_idx][0] = pillarSM[pillar_idx_inBlock][point_idx].x;
      pillarOutSM[pillar_idx_inBlock][point_idx][1] = pillarSM[pillar_idx_inBlock][point_idx].y;
      pillarOutSM[pillar_idx_inBlock][point_idx][2] = pillarSM[pillar_idx_inBlock][point_idx].z;
      pillarOutSM[pillar_idx_inBlock][point_idx][3] = pillarSM[pillar_idx_inBlock][point_idx].w;

      pillarOutSM[pillar_idx_inBlock][point_idx][4] = mean.x;
      pillarOutSM[pillar_idx_inBlock][point_idx][5] = mean.y;
      pillarOutSM[pillar_idx_inBlock][point_idx][6] = mean.z;

      pillarOutSM[pillar_idx_inBlock][point_idx][7] = center.x;
      pillarOutSM[pillar_idx_inBlock][point_idx][8] = center.y;
      pillarOutSM[pillar_idx_inBlock][point_idx][9] = center.z;

    } else {
      pillarOutSM[pillar_idx_inBlock][point_idx][0] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][1] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][2] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][3] = 0;

      pillarOutSM[pillar_idx_inBlock][point_idx][4] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][5] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][6] = 0;

      pillarOutSM[pillar_idx_inBlock][point_idx][7] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][8] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][9] = 0;
    }

    __syncthreads();

    for(int i = 0; i < FEATURES_SIZE; i ++) {
      int outputSMId = pillar_idx_inBlock*WARP_SIZE*FEATURES_SIZE + i* WARP_SIZE + point_idx;
      int outputId = pillar_idx*WARP_SIZE*FEATURES_SIZE + i* WARP_SIZE + point_idx;
      features[outputId] = ((float*)pillarOutSM)[outputSMId] ;
    }

}

hipError_t generateFeatures_launch(float* voxel_features,
    float* voxel_num_points,
    float* coords,
    unsigned int *params,
    float voxel_x, float voxel_y, float voxel_z,
    float range_min_x, float range_min_y, float range_min_z,
    float* features,
    hipStream_t stream)
{
    dim3 blocks( (MAX_VOXELS+WARPS_PER_BLOCK-1)/WARPS_PER_BLOCK);
    dim3 threads(WARPS_PER_BLOCK*WARP_SIZE);

    generateFeatures_kernel<<<blocks, threads, 0, stream>>>
      (voxel_features,
      voxel_num_points,
      coords,
      params,
      voxel_x, voxel_y, voxel_z,
      range_min_x, range_min_y, range_min_z,
      features);

    hipError_t err = hipGetLastError();
    return err;
}
//generateFeatures>>

